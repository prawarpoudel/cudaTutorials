
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

using namespace std;

void createMatrix(int* myMat,int row,int col)
{
	for(int i=0;i<row;i++)
	{
		for(int j=0;j<col;j++)
		{
			myMat[i*col+j] = rand()%10;
		}
	}
}

void printMatrix(int* myMat,int row,int col)
{
	for(int i=0;i<row;i++)
	{
		for(int j=0;j<col;j++)
		{
			cout<<myMat[i*col+j]<<" ";
		}
		cout<<endl;
	}
}

__global__
void multiplyMatrix(int* matA,int* matB,int* resultMat,int rowA,int rowB,int colB)
{
	int i  = threadIdx.x;
	for(int j=0;j<colB;j++)
	{
		int sum = 0;
		for(int k=0;k<rowB;k++)
		{
			sum+=(matA[i*rowB+k]*matB[k*colB+j]);
		}
		resultMat[i*colB+j] = sum;
	}
}

void multiplyMatrixSerial(int* matA, int* matB, int* resultMat, int rowA, int rowB, int colB)
{
	for (int i = 0; i<rowA; i++)
	{
		for (int j = 0; j<colB; j++)
		{
			int sum = 0;
			for (int k = 0; k<rowB; k++)
			{
				sum += (matA[i*rowB + k] * matB[k*colB + j]);
			}
			resultMat[i*colB + j] = sum;
		}
	}
}

int main()
{
	int rowA = 3;
	int rowB = 4;
	int colB = 3;

	//allocate memory in host
	int *matA = new int[rowA*rowB*sizeof(int)];
	int *matB = new int[rowB*colB*sizeof(int)];
	int *matC = new int[rowA*colB*sizeof(int)];

	//allocate memory in device
	int *dA, *dB, *dC;
	hipMalloc((void**)&dA,rowA*rowB*sizeof(int));
	hipMalloc((void**)&dB,rowB*colB*sizeof(int));
	hipMalloc((void**)&dC,rowA*colB*sizeof(int));

	cout<<"Creating matrix..."<<endl;
	createMatrix(matA,rowA,rowB);
	createMatrix(matB,rowB,colB);
	cout<<"Creating matrix completed"<<endl;

	//copy from host to device
	hipMemcpy(dA,matA,rowA*rowB*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dB,matB,rowB*colB*sizeof(int),hipMemcpyHostToDevice);

	cout<<"MatrixA: "<<endl;
	printMatrix(matA,rowA,rowB);
	cout<<"MatrixB: "<<endl;
	printMatrix(matB,rowB,colB);

	//each thread will compute a row of elements in result matrix
	multiplyMatrix <<<1,rowA>>> (dA,dB,dC,rowA,rowB,colB);

	//copy result from device to host
	hipMemcpy(matC,dC,rowA*colB*sizeof(int),hipMemcpyDeviceToHost);

	cout<<"The parallel result matrix is: "<<endl;
	printMatrix(matC,rowA,colB);

	multiplyMatrixSerial(matA,matB,matC,rowA,rowB,colB);
	cout << "The serial result matrix is: " << endl;
	printMatrix(matC, rowA, colB);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	delete[] matA;
	delete[] matB;
	delete[] matC;

	return 0;
}